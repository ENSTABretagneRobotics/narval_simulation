#include "hip/hip_runtime.h"
#include "optix_acoustics.h"

#include <rtac_optix/helpers/maths.h>

extern "C" {
    __constant__ Params params;
}

/**
 * This the "main" program of ray tracing. It is called for each ray. 
 */
extern "C" __global__ void __raygen__acoustic_ray()
{
    auto idx = optixGetLaunchIndex();
    
    AcousticRay ray;
    ray.value          = Complex<float>(0,0);
    ray.travelDistance = 0.0f;
    ray.travelTime     = 0.0f;

    for(int i = 0; i < params.iterationCount; i++) {
        ray.trace(params.topObject,
                  params.rayOrigins[idx.x],
                  params.rayDirections[idx.x],
                  1.0e-3, params.rayStep);
        params.rayPaths[params.iterationCount*idx.x + i] = params.rayOrigins[idx.x];
    }
}

/**
 * The miss program is called if a ray does not intersected with anything
 * before reaching its maximumrange.
 *
 * Here it is used to implement a Euler intgration scheme to curve the rays. At
 * each step, the local sound celerity and its gradiant is fetched from the
 * local Acoustic medium and are used to compute a change in ray direction.
 */
extern "C" __global__ void __miss__acoustic_ray()
{
    // This get the propagation medium information from the shader binding
    // table.

    auto medium = (const AcousticMedium*)optixGetSbtDataPointer();
    auto idx    = optixGetLaunchIndex();
    
    // These fetch current ray information (payload form ray registers, ray
    // origin in world frame and direction in workd frame.
    auto ray      = AcousticRay::from_registers();
    float3 origin = AcousticRay::world_origin();
    float3 dir    = AcousticRay::world_direction();
    
    // Euler integration step
    float3 deltaDir = medium->profile.get_direction_delta(origin, dir);
    params.rayOrigins[idx.x]    = origin + params.rayStep * dir;
    params.rayDirections[idx.x] = normalized(dir + params.rayStep * deltaDir);

    // updating ray payload
    float dist = params.rayStep*length(dir);
    ray.travelDistance += dist;
    ray.travelTime     += dist / medium->profile.get_celerity(origin.z);
    ray.set_payload(); // storing ray payload in ray registers
}

/**
 * The closest hit program is called on the ray closest intersection with an
 * object.
 *
 * This specific program implements a perfect mirror material (suitable for
 * water surface reflexion for example)
 */
extern "C" __global__ void __closesthit__mirror()
{
    auto idx = optixGetLaunchIndex();

    // is a perfect mirror material for now
    float3 hitPosition, hitNormal;
    helpers::get_triangle_hit_data(hitPosition, hitNormal);

    hitPosition = optixTransformPointFromObjectToWorldSpace(hitPosition);
    hitNormal   = optixTransformNormalFromObjectToWorldSpace(hitNormal);

    float3 origin = AcousticRay::world_origin();
    float3 dir    = AcousticRay::world_direction();

    float3 nextDir = normalized(dir - 2.0*dot(dir, hitNormal)*hitNormal);

    auto ray = AcousticRay::from_registers();
    params.rayOrigins[idx.x]    = hitPosition + 0.05 * nextDir;
    params.rayDirections[idx.x] = nextDir;

    ray.set_payload();
}


